#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>

constexpr int			   NM = 5;				// Number of Investigated Modes
constexpr double Perturbation = 1e-3;

#include "SphericalStability_SystemDefinition.cuh"
#include "MPGOS/SingleSystem_PerThread.cuh"

#define PI 3.14159265358979323846

using namespace std;

// Physical control parameters
const int NumberOfFrequency1 = 101;
const int NumberOfFrequency2 = 101;
const int NumberOfAmplitude1 = 2;
const int NumberOfAmplitude2 = 2;

// Solver Configuration
#define SOLVER RKCK45 // RK4, RKCK45
const int NT	= NumberOfFrequency1 * NumberOfFrequency2; // NumberOfThreads
const int SD	= 2+2*NM;// SystemDimension
const int NCP	= 23;    // NumberOfControlParameters
const int NSP	= 5*NM;	 // NumberOfSharedParameters
const int NISP	= 0;     // NumberOfIntegerSharedParameters
const int NE	= 1;     // NumberOfEvents
const int NA	= 2;     // NumberOfAccessories
const int NIA	= 0;     // NumberOfIntegerAccessories
const int NDO	= 0;     // NumberOfPointsOfDenseOutput

void Linspace(vector<double>&, double, double, int);
void Logspace(vector<double>&, double, double, int);
void FillSolverObject(ProblemSolver<NT, SD, NCP, NSP, NISP, NE, NA, NIA, NDO, SOLVER, double>&, const vector<double>&, const vector<double>&, const vector<double>&, const vector<double>&, int, int);
void PerturbateSolverObject(ProblemSolver<NT, SD, NCP, NSP, NISP, NE, NA, NIA, NDO, SOLVER, double>&, int);

int main()
{

	int BlockSize = 64;

	vector<double> Frequency1(NumberOfFrequency1, 0);
	vector<double> Frequency2(NumberOfFrequency2, 0);
	vector<double> Amplitude1(NumberOfAmplitude1, 0);
	vector<double> Amplitude2(NumberOfAmplitude2, 0);

	Logspace(Frequency1, 20.0, 2000.0, NumberOfFrequency1);
	Logspace(Frequency2, 20.0, 2000.0, NumberOfFrequency2);
	Linspace(Amplitude1, 0.0, 2.0, NumberOfAmplitude1);
	Linspace(Amplitude2, 0.0, 2.0, NumberOfAmplitude2);

	// Setup CUDA a device
	ListCUDADevices();

	int MajorRevision = 3;
	int MinorRevision = 5;
	int SelectedDevice = SelectDeviceByClosestRevision(MajorRevision, MinorRevision);

	PrintPropertiesOfSpecificDevice(SelectedDevice);

	// Solver Object configuration
	int NumberOfProblems = NumberOfFrequency1 * NumberOfFrequency2 * NumberOfAmplitude1 * NumberOfAmplitude2;
	int NumberOfThreads = NT;

	ProblemSolver<NT, SD, NCP, NSP, NISP, NE, NA, NIA, NDO, SOLVER, double> CheckSphericalStability(SelectedDevice);

	CheckSphericalStability.SolverOption(ThreadsPerBlock, BlockSize);
	CheckSphericalStability.SolverOption(RelativeTolerance, 0, 1e-12);
	CheckSphericalStability.SolverOption(AbsoluteTolerance, 1, 1e-12);
	for (int i = 2; i < SD; i++)	// Set tolerance of Surface Dynamics
	{
		CheckSphericalStability.SolverOption(RelativeTolerance, 0, 1e-12);
		CheckSphericalStability.SolverOption(AbsoluteTolerance, 1, 1e-12);
	}
	CheckSphericalStability.SolverOption(EventDirection, 0, -1);
	CheckSphericalStability.SolverOption(EventStopCounter, 0, 1);
	CheckSphericalStability.SolverOption(DenseOutputTimeStep, -1e-2);

	// SIMULATIONS ------------------------------------------------------------------------------------

	int NumberOfSimulationLaunches = NumberOfProblems / NumberOfThreads;
	int ProblemStartIndex;

	vector< vector<double>> CollectedData;
	CollectedData.resize(NumberOfThreads, vector<double>(NM + 10, 0));
	// 6 physical paramters +
	// 1 initial time of the stability analysis
	// 1 end time of the stability analysis
	// 2 state varialbes (x1, x2) before the stability analysis
	// NM growth rates for each investiageted modes

	double ActualPA1;
	double ActualPA2;
	clock_t SimulationStart = clock();
	for (int LaunchCounter = 1; LaunchCounter < NumberOfSimulationLaunches; LaunchCounter++)
	{
		// Fill Solver Object
		ProblemStartIndex = LaunchCounter * NumberOfThreads;
		FillSolverObject(CheckSphericalStability, Frequency1, Frequency2, Amplitude1, Amplitude2, ProblemStartIndex, NumberOfThreads);
		CheckSphericalStability.SynchroniseFromHostToDevice(All);

		// Generate a unique filename for the current launch
		stringstream StreamFilename;
		StreamFilename.precision(2);
		StreamFilename.setf(ios::fixed);

		ActualPA1 = CheckSphericalStability.GetHost(0, ControlParameters, 15);
		ActualPA2 = CheckSphericalStability.GetHost(0, ControlParameters, 17);
		StreamFilename << "SphericalStability_PA1_" << ActualPA1 << "_PA2_" << ActualPA2 << ".txt";

		string Filename = StreamFilename.str();
		remove(Filename.c_str());

		// Collect physical parameters
		for (int tid = 0; tid < NumberOfThreads; tid++)
		{
			CollectedData[tid][0] = CheckSphericalStability.GetHost(tid, ControlParameters, 15);
			CollectedData[tid][1] = CheckSphericalStability.GetHost(tid, ControlParameters, 16);
			CollectedData[tid][2] = CheckSphericalStability.GetHost(tid, ControlParameters, 17);
			CollectedData[tid][3] = CheckSphericalStability.GetHost(tid, ControlParameters, 18);
			CollectedData[tid][4] = CheckSphericalStability.GetHost(tid, ControlParameters, 19);
			CollectedData[tid][5] = CheckSphericalStability.GetHost(tid, ControlParameters, 20);
		}
		cout << "LaunchCounter: " << LaunchCounter << "/" << NumberOfSimulationLaunches - 1 << " ... ";
		cout << "Transient Iterations ... ";
		for (int i = 0; i < 1024; i++)
		{
			//cout << "Transient Iteration: " << i << endl;
			CheckSphericalStability.Solve();
			CheckSphericalStability.InsertSynchronisationPoint();
			CheckSphericalStability.SynchroniseSolver();
		}
	
		// Collect date aifter the treandient iteration
		CheckSphericalStability.SynchroniseFromDeviceToHost(TimeDomain);
		CheckSphericalStability.SynchroniseFromDeviceToHost(Accessories);
		CheckSphericalStability.InsertSynchronisationPoint();
		CheckSphericalStability.SynchroniseSolver();
		for (int tid = 0; tid < NumberOfThreads; tid++)
		{
			CollectedData[tid][6] = CheckSphericalStability.GetHost(tid, TimeDomain, 0);
			CollectedData[tid][8] = CheckSphericalStability.GetHost(tid, Accessories, 0);
			CollectedData[tid][9] = CheckSphericalStability.GetHost(tid, Accessories, 1);
		}
			
		// Stability analysis and data collection
		PerturbateSolverObject(CheckSphericalStability, NumberOfThreads);
		cout << "Stability Iterations ... ";
		for (int i = 0; i < 32; i++)
		{
			//cout << "Stability Iteration: " << i << endl;
			CheckSphericalStability.Solve();
			CheckSphericalStability.InsertSynchronisationPoint();
			CheckSphericalStability.SynchroniseSolver();
		}

		CheckSphericalStability.SynchroniseFromDeviceToHost(All);
		CheckSphericalStability.InsertSynchronisationPoint();
		CheckSphericalStability.SynchroniseSolver();
		cout << "Done" << endl;
		for (int tid = 0; tid < NumberOfThreads; tid++)
		{
			CollectedData[tid][7] = CheckSphericalStability.GetHost(tid, TimeDomain, 0);

			// Calculate Growth rates
			for (int i = 0; i < NM; i++)
			{
				CollectedData[tid][10 + i] = log(abs(CheckSphericalStability.GetHost(tid, ActualState, i + 2)) / Perturbation) / (CollectedData[tid][7] - CollectedData[tid][6]);
			}
		}
		
		// Save collected data to file
		ofstream DataFile;
		DataFile.open(Filename.c_str(), std::fstream::app);
		int Width = 18;
		DataFile.precision(10);
		DataFile.flags(ios::scientific);

		for (int tid = 0; tid < NumberOfThreads; tid++)
		{
			for (int col = 0; col < 10 + NM; col++)
			{
				if (col < (10 + NM - 1))
				{
					DataFile.width(Width); DataFile << CollectedData[tid][col] << ',';
				}
				else
				{
					DataFile.width(Width); DataFile << CollectedData[tid][col];
				}
			}
			DataFile << '\n';
		}
		DataFile.close();
	}
	clock_t SimulationEnd = clock();
		cout << "Total simulation time: " << 1000.0*(SimulationEnd - SimulationStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
	return 0;
}


// ------------------------------------------------------------------------------------------------

void Linspace(vector<double>& x, double B, double E, int N)
{
	double Increment;

	x[0] = B;

	if (N > 1)
	{
		x[N - 1] = E;
		Increment = (E - B) / (N - 1);

		for (int i = 1; i < N - 1; i++)
		{
			x[i] = B + i * Increment;
		}
	}
}

void Logspace(vector<double>& x, double B, double E, int N)
{
	x[0] = B;

	if (N > 1)
	{
		x[N - 1] = E;
		double ExpB = log10(B);
		double ExpE = log10(E);
		double ExpIncr = (ExpE - ExpB) / (N - 1);
		for (int i = 1; i < N - 1; i++)
		{
			x[i] = pow(10, ExpB + i * ExpIncr);
		}
	}
}

// ------------------------------------------------------------------------------------------------
void FillSolverObject(ProblemSolver<NT, SD, NCP, NSP, NISP, NE, NA, NIA, NDO, SOLVER, double>& Solver, const vector<double>& F1_Values, const vector<double>& F2_Values, const vector<double>& PA1_Values, const vector<double>& PA2_Values, int ProblemStartIndex, int NumberOfThreads)
{
	// Declaration of physical control parameters
	double P1; // pressure amplitude1 [bar]
	double P2; // frequency1          [kHz]
	double P3; // pressure amplitude2 [bar]
	double P4; // frequency2          [kHz]

	// Declaration of constant parameters
	double P5 = 0.0*PI;	// phase shift          [-]
	double P6 = 10.0;	// equilibrium radius   [mum]
	double P7 = 1.0;	// ambient pressure     [bar]
	double P9 = 1.4;	// polytrophic exponent [-]

	// Material properties
	double Pv = 3.166775638952003e+03;
	double Rho = 9.970639504998557e+02;
	double ST = 0.071977583160056;
	double Vis = 8.902125058209557e-04;
	double CL = 1.497251785455527e+03;

	// Auxiliary variables
	double Pinf;
	double PA1;
	double PA2;
	double RE;
	double f1;
	double f2;

	// Set Shared Parameters
	double n = 2.0;
	for (int i = 0; i < NM; i++)
	{
		Solver.SetHost(SharedParameters, 0 + i * 5, n);
		Solver.SetHost(SharedParameters, 1 + i * 5, n - 1);
		Solver.SetHost(SharedParameters, 2 + i * 5, n * (n + 2)*(n + 2));
		Solver.SetHost(SharedParameters, 3 + i * 5, n * (n - 1)*(n + 2));
		Solver.SetHost(SharedParameters, 4 + i * 5, (n - 1)*(n + 1)*(n + 2));
		n +=1.0;
	}

	int ProblemNumber = 0;
	int GlobalCounter = 0;
	for (auto const& CP4 : PA2_Values) // pressure amplitude2 [bar]
	{
		for (auto const& CP3 : PA1_Values) // pressure amplitude1 [bar]
		{
			for (auto const& CP2 : F2_Values) // frequency2 [kHz]
			{
				for (auto const& CP1 : F1_Values) // frequency1 [kHz]
				{
					if (GlobalCounter < ProblemStartIndex)
					{
						GlobalCounter++;
						continue;
					}

					// Update physical parameters
					P1 = CP3;
					P2 = CP1;
					P3 = CP4;
					P4 = CP2;

					Solver.SetHost(ProblemNumber, TimeDomain, 0, 0);
					Solver.SetHost(ProblemNumber, TimeDomain, 1, 1e10);

					// Initial conditions are the equilibrium condition y1=1; y2=0;
					Solver.SetHost(ProblemNumber, ActualState, 0, 1.0);
					Solver.SetHost(ProblemNumber, ActualState, 1, 0.0);

					// BLA - Initial Perturbation
					for (int i = 2; i < 2 + 2 * NM; i++)
					{
						Solver.SetHost(ProblemNumber, ActualState, i, 0.0);
					}

					// Scaling of physical parameters to SI
					Pinf = P7 * 1e5;
					PA1	 = P1 * 1e5;
					PA2	 = P3 * 1e5;
					RE	 = P6 / 1e6;

					// Scale to angular frequency
					f1 = 2.0*PI*(P2 * 1000);
					f2 = 2.0*PI*(P4 * 1000);

					// System coefficients and other, auxiliary parameters
					Solver.SetHost(ProblemNumber, ControlParameters, 0, (2.0*ST / RE + Pinf - Pv) * pow(2.0*PI / RE / f1, 2.0) / Rho);
					Solver.SetHost(ProblemNumber, ControlParameters, 1, (1.0 - 3.0*P9) * (2 * ST / RE + Pinf - Pv) * (2.0*PI / RE / f1) / CL / Rho);
					Solver.SetHost(ProblemNumber, ControlParameters, 2, (Pinf - Pv) * pow(2.0*PI / RE / f1, 2.0) / Rho);
					Solver.SetHost(ProblemNumber, ControlParameters, 3, (2.0*ST / RE / Rho) * pow(2.0*PI / RE / f1, 2.0));
					Solver.SetHost(ProblemNumber, ControlParameters, 4, (4.0*Vis / Rho / pow(RE, 2.0)) * (2.0*PI / f1));
					Solver.SetHost(ProblemNumber, ControlParameters, 5, PA1 * pow(2.0*PI / RE / f1, 2.0) / Rho);
					Solver.SetHost(ProblemNumber, ControlParameters, 6, PA2 * pow(2.0*PI / RE / f1, 2.0) / Rho);
					Solver.SetHost(ProblemNumber, ControlParameters, 7, (RE*f1*PA1 / Rho / CL) * pow(2.0*PI / RE / f1, 2.0));
					Solver.SetHost(ProblemNumber, ControlParameters, 8, (RE*f2*PA2 / Rho / CL) * pow(2.0*PI / RE / f1, 2.0));
					Solver.SetHost(ProblemNumber, ControlParameters, 9, RE*f1 / (2.0*PI) / CL);
					Solver.SetHost(ProblemNumber, ControlParameters, 10, 3.0*P9);
					Solver.SetHost(ProblemNumber, ControlParameters, 11, P4 / P2);
					Solver.SetHost(ProblemNumber, ControlParameters, 12, P5);

					Solver.SetHost(ProblemNumber, ControlParameters, 13, 2.0*PI / f1);	// tref
					Solver.SetHost(ProblemNumber, ControlParameters, 14, RE);			// Rref

					Solver.SetHost(ProblemNumber, ControlParameters, 15, P1);
					Solver.SetHost(ProblemNumber, ControlParameters, 16, P2);
					Solver.SetHost(ProblemNumber, ControlParameters, 17, P3);
					Solver.SetHost(ProblemNumber, ControlParameters, 18, P4);
					Solver.SetHost(ProblemNumber, ControlParameters, 19, P5);
					Solver.SetHost(ProblemNumber, ControlParameters, 20, P6);

					Solver.SetHost(ProblemNumber, ControlParameters, 21, PA1 != 0 ? sqrt(Vis / Rho / f1) : 0.0);
					Solver.SetHost(ProblemNumber, ControlParameters, 22, PA2 != 0 ? sqrt(Vis / Rho / f2) : 0.0);

					ProblemNumber++;

					if (ProblemNumber == NumberOfThreads)
						goto ExitSolverFilling;
				}
			}
		}
	}
	ExitSolverFilling:;
}

void PerturbateSolverObject(ProblemSolver<NT, SD, NCP, NSP, NISP, NE, NA, NIA, NDO, SOLVER, double>& Solver, int NumberOfThreads)
{
	Solver.SynchroniseFromDeviceToHost(ActualState);
	Solver.InsertSynchronisationPoint();
	Solver.SynchroniseSolver();
	int ProblemNumber = 0;
	while (ProblemNumber < NumberOfThreads)
	{
		for (int i = 2; i < 2 + 2 * NM; i++)
		{
			if (i < 2 + NM) { Solver.SetHost(ProblemNumber, ActualState, i, Perturbation); }
			else			{ Solver.SetHost(ProblemNumber, ActualState, i, 0.0);  }
		}
		ProblemNumber++;
	}
	Solver.SynchroniseFromHostToDevice(ActualState);
	Solver.InsertSynchronisationPoint();
	Solver.SynchroniseSolver();
}